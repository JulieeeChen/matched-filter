#include "hip/hip_runtime.h"
//============================================================================
// Name        : MF6.cpp
// Author      : Sohrab
// Version     : 1
// Copyright   : Hi!
// Description : Matched Filter in C++, Ansi-style
//============================================================================

#include <iostream>
#include <string>
#include <cmath>
#include <math.h>
#include <ctime>
#include <complex>
#include <vector>
#include <string>
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#include <thrust/complex.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// references
#define date_ref "1015"
#define obj_ref "1"
#define run_ref 2

// 0: down sampling,  1: averaging, 2: nothing
#define average 2

//internal distance
#define int_dst 2.615

// TX relative position to TX's starting point
#define Tx_pos_x 0.41
#define Tx_pos_y -0.028
#define Tx_pos_z -0.012

//starting point of using samples
#define N_spl_fr 1000
#define N_lfreq_spl  0
#define N_hfreq_spl  0
#define N_mfreq_spl (N_spl_fr/2)-N_lfreq_spl-N_hfreq_spl
#define N_mfreq_spl_slow 2*((N_spl_fr/2)-N_lfreq_spl-N_hfreq_spl)

// Number of frames for each axis
#define N_x_stg 20 //1667
#define N_z_stg 20

//constants
#define Ts 6e-3
#define Rs 5e5
#define lambda 4.983873e-3

//step size between each two frames considered
#define dlx 0.005 * 0.006
#define dlz lambda/2
#define linxmax -dlx/2-(N_x_stg-1)*dlx
#define linzmax -dlz/2-(N_z_stg-1)*dlz

// environment dimensions
#define xmin -5
#define xmax 5


#define ymin 0
#define ymax 10


#define zmin -1.5
#define zmax 1.5


//resolution
#define res 0.07

//scientific values for some constants
#define sci_fac 1e8

#define c_sci 2.9979
#define fc_sci 609
#define As1_sci 1.5001e4
#define As2_sci 7.5005e3


#define file_size 100020000
#define size1 3000

#define BLOCK_WIDTH 8

#define Beat_I(uu, vv, nn) Beat_I[uu*N_z_stg*size1 + vv*size1 + nn]
#define Beat_R(uu, vv, nn) Beat_R[uu*N_z_stg*size1 + vv*size1 + nn]
#define cell_MF(xx, yy, zz) cell_MF[xx*Ny*Nz + yy*Nz + zz]
#define deviceCellMF(xx, yy, zz) deviceCellMF[xx*Ny*Nz + yy*Nz + zz]
using namespace std;

/****************** FUNCTIONS ******************/

struct indices {
	int kx;
	int ky;
	int kz;
};

indices idxfinder(int n1, int n2, int n3, int k) {

	k = k % (n1 * n2 * n3);

	indices I;
	I.kx = k % n1;
	I.ky = ((int) floor(k / n1)) % n2;
	I.kz = ((int) floor(k / (n1 * n2))) % n3;

	return I;
}

/****************************************************/

/************* KERNEL CALL *************************/

__global__ void matchedFilterKernel(float* Beat_R, float* Beat_I, thrust::complex<float>* cell_MF, int Nx, int Ny, int Nz) {

    #define MF_x_axis(xx) (xx*res + xmin)
    #define MF_y_axis(yy) (yy*res + ymin)
    #define MF_z_axis(zz) (zz*res + zmin)
    #define u_axis(uu) (-dlx/2 - uu*dlx)
    #define v_axis(vv) (-dlz/2 - vv*dlz)

    const float pi = acosf(-1);
	const thrust::complex<double> i(0, 1);
	const thrust::complex<float> i_float(0, 1);

    int xx, yy, zz;
    xx = blockIdx.x * blockDim.x + threadIdx.x;
    yy = blockIdx.y * blockDim.y + threadIdx.y;
    zz = blockIdx.z * blockDim.z + threadIdx.z;

    if(xx < Nx && yy < Ny && zz < Nz) {
        float cell_z = MF_z_axis(zz);
        float cell_y = MF_y_axis(yy);
        float cell_x = MF_x_axis(xx);

        thrust::complex<float> cell_sum = 0;

        // for(int nn = 0; nn < size1; nn++)  // 3000
        //             Beat[nn] = Beat_R(uu, vv, nn) + i_float * Beat_I(uu, vv, nn);
        // __shared__ complex<float> Beat[size1]


        float cell_dist_t = sqrtf(
                (cell_x - Tx_pos_x) * (cell_x - Tx_pos_x)
                        + (cell_y - Tx_pos_y) * (cell_y - Tx_pos_y)
                        + (cell_z - Tx_pos_z) * (cell_z - Tx_pos_z));


        for (int uu = 0; uu < N_x_stg; uu++) { // N_x_stg  

            float x_diff = (cell_x - u_axis(uu)) * (cell_x - u_axis(uu));

            for (int vv = 0; vv < N_z_stg; vv++) { // 2d receiver 1667*20

                float temp_tau = (cell_dist_t + int_dst * 2 + sqrtf( x_diff +
                        (cell_z - v_axis(vv)) * (cell_z - v_axis(vv)) + cell_y * cell_y) ) / c_sci;

                thrust::complex<float> temp_sig = exp(-i_float * (float) fmod((float)2.0 * pi * fc_sci * temp_tau, 2*pi) );
                thrust::complex<float> Beat[size1];


                thrust::complex<float> cell_sig_fst_temp[N_mfreq_spl];
                thrust::complex<float> cell_sig_slow_temp[N_mfreq_spl_slow];

                for(int nn = 0; nn < size1; nn++)  // 3000
                    Beat[nn] = Beat_R(uu, vv, nn) + i_float * Beat_I(uu, vv, nn);



                for (int nn = 0; nn < N_mfreq_spl; nn++) { // for each fixed receiver and object location, 3000 samples
                    cell_sig_fst_temp[nn] = temp_sig * exp(-i_float * (float) fmod((float)(2.0 * pi *
                            As1_sci * (N_lfreq_spl / Rs + nn / Rs) * temp_tau), 2*pi));
                    cell_sum += cell_sig_fst_temp[nn] * (Beat_R(uu, vv, nn) + i_float * Beat_I(uu, vv, nn)); //Beat[nn];
                }


                for (int nn = N_mfreq_spl; nn < 2*N_mfreq_spl; nn++) {
                    cell_sum += cell_sig_fst_temp[2*N_mfreq_spl-1-nn] * Beat[nn];
                }



                for (int nn = 0; nn < N_mfreq_spl_slow; nn++) {
                    cell_sig_slow_temp[nn] = temp_sig * exp(-i_float * (float) fmod((float)(2.0 * pi
                            * As2_sci * (N_lfreq_spl * 2 / Rs + nn / Rs) * temp_tau), 2*pi) );
                    cell_sum += cell_sig_slow_temp[nn] * Beat[nn+2*N_mfreq_spl];
                }

                for (int nn = N_mfreq_spl_slow; nn < 2* N_mfreq_spl_slow; nn++) {
                    cell_sum += cell_sig_slow_temp[2*N_mfreq_spl_slow-1-nn] * Beat[nn+2*N_mfreq_spl];
                }


            }



        }

        cell_MF(xx, yy, zz) = cell_sum;
    }


    #undef MF_x_axis
    #undef MF_y_axis
    #undef MF_z_axis
    #undef u_axis
    #undef v_axis

}


/**************************************************/




int
main(void)
{
    hipError_t err = hipSuccess;
    /************* LARGE ARRAY DECLRATATIONS AND NX, NY, NZ************/
    int Nx = 143; // (int) floor((xmax-xmin)/res)+1; //143
    int Ny = 10; //(int) floor((ymax-ymin)/res)+1; //143
    int Nz = 10; //(int) floor((zmax-zmin)/res)+1; //43

    // complex<float> cell_sig_fst[N_x_stg][N_z_stg][N_mfreq_spl];
    // complex<float> cell_sig_slow[N_x_stg][N_z_stg][N_mfreq_spl_slow];

    // Allocate host memory

    float* Beat_R = (float *)malloc(N_x_stg * N_z_stg * size1 * sizeof(float)); //[N_x_stg][N_z_stg][size1] = {};
    float* Beat_I = (float *)malloc(N_x_stg * N_z_stg * size1 * sizeof(float)); //[N_x_stg][N_z_stg][size1] = {};
    thrust::complex<float>* cell_MF = (thrust::complex<float>*)malloc(Nx * Ny * Nz * sizeof(thrust::complex<float>)); //[Nx][Ny][Nz] 143 * 143 *43
    
    // Verify that allocations succeeded
    if (Beat_R == NULL || Beat_I == NULL || cell_MF == NULL )
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    /**************************************************/

	clock_t begin = clock();
    clock_t end;
	// srand (time(NULL));
	// for (int ii = 0; ii < N_z_stg; ii++)
	// 	for (int jj = 0; jj < N_x_stg; jj++)
	// 		for (int kk = 0; kk < size1; kk++) {
	// 			Beat_R(jj, ii, kk) = (rand()%10 + 1)/10;
	// 			Beat_I(jj, ii, kk) = (rand()%10 + 1)/10;
    // 		}
    

	/*********** READ THE .BIN FILES ************/
	FILE *fp = fopen("/home/synrg-gpu1/Desktop/MF6/testReal.bin","rb");


	for (int ii = 0; ii < N_z_stg; ii++){
		for (int jj = 0; jj < N_x_stg; jj++) {

			float b[size1];
			fseek(fp, (ii*N_x_stg + jj)*size1*4, SEEK_SET);
			fread(b, sizeof *b, size1, fp);
			for(int kk = 0; kk < size1; kk++) {
				Beat_R(jj, ii, kk) = 1;
				//if (ii == 0 && jj == 1 && kk < 500) cout << b[kk] << endl;
			}
		}
	}

	fclose(fp);

	cout << "Successfully read the file in " << (double) (clock() - begin) / CLOCKS_PER_SEC << " seconds!" << endl;

	FILE *fp2 = fopen("/home/synrg-gpu1/Desktop/MF6/testImag.bin","rb");


	for (int ii = 0; ii < N_z_stg; ii++){
		for (int jj = 0; jj < N_x_stg; jj++) {

			float b[size1];
			fseek(fp2, (ii*N_x_stg + jj)*size1*4, SEEK_SET);
			fread(b, sizeof *b, size1, fp2);

			for(int kk = 0; kk < size1; kk++) {
				Beat_I(jj, ii, kk) = 0;

			}
		}
	}

	fclose(fp2);
	cout << "Successfully read the files in " << (double) (clock() - begin) / CLOCKS_PER_SEC << " seconds!" << endl;
	// cout << Beat_I(149, 14, 149)<< endl << endl;

	/******************** END OF READ FILE *********************/

	//some constants
	const float pi = acos(-1);
	const thrust::complex<double> i(0, 1);
    const thrust::complex<float> i_float(0, 1);


    for (int i = 0; i < 1; i++){
        for (int j = 0; j < 1; j++){
            for (int k = 0; k < 10; k++) {
                cout << cell_MF(k, j, i) << " ";
            }
            std::endl( std::cout );
        }
       std::endl( std::cout );
  }

    float* deviceBeatI;
    float* deviceBeatR;
    thrust::complex<float>* deviceCellMF;

    clock_t begin_mem = clock();
    
    // Allocate GPU memory
    err = hipMalloc((void **) &deviceBeatR , N_z_stg * N_x_stg * size1 * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceBeatR (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **) &deviceBeatI , N_z_stg * N_x_stg * size1 * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceBeatI (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **) &deviceCellMF , Nx * Ny * Nz * sizeof(thrust::complex<float>));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceCellMF (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    printf("Copy input data from the host memory to the CUDA device\n");

    err = hipMemcpy(deviceBeatR, Beat_R, N_z_stg * N_x_stg * size1 * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy deviceBeatR from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(deviceBeatI, Beat_I, N_z_stg * N_x_stg * size1 * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy deviceBeatI from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    cout << "hi!" << endl;

    
    dim3 DimGrid(ceil(Nx * 1.0 / BLOCK_WIDTH), ceil(Ny * 1.0 / BLOCK_WIDTH), ceil(Nz * 1.0 /BLOCK_WIDTH));
    dim3 DimBlock(BLOCK_WIDTH, BLOCK_WIDTH, BLOCK_WIDTH);

    cout << "Allocating & copying memory DONE! Time taken:" << (double) (clock() - begin_mem) / CLOCKS_PER_SEC;
    
    matchedFilterKernel<<<DimGrid, DimBlock>>>(deviceBeatR, deviceBeatI, deviceCellMF, Nx, Ny, Nz);
    hipDeviceSynchronize();
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matchedFilterKernel  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    begin_mem = clock();

    printf("Copy output data from the CUDA device to the host memory\n");

    err = hipMemcpy(cell_MF, deviceCellMF, Nx * Ny * Nz * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy deviceCellMF from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    cout << "Copying memory back DONE! Time taken:" << (double) (clock() - begin_mem) / CLOCKS_PER_SEC;


	cout << "Hi! \n";
	cout << Nx << endl;
	cout << Ny << endl;
	cout << Nz << endl;
	cout << N_x_stg << endl;
	cout << N_z_stg << endl;
	cout << N_mfreq_spl_slow << endl;

	end = clock();
    cout << "DONE! Time taken:" << (double) (end - begin) / CLOCKS_PER_SEC;

    for (int i = 0; i < 1; i++){
        for (int j = 0; j < 5; j++){
            for (int k = 0; k < 5; k++) {
                cout << cell_MF(k, j, i) << " ";
            }
            std::endl( std::cout );
        }
        std::endl( std::cout );
    }

    hipFree(deviceBeatR);
    hipFree(deviceBeatI);
    hipFree(deviceCellMF);

    free(Beat_R);
    free(Beat_I);
    free(cell_MF);

	return 0;
}
